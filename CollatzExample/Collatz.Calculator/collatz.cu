
#include <hip/hip_runtime.h>

extern "C" __global__ void collatz(int start, int count, int* stepsArray, int max) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < count) {
		int n = start + i;

		for (int steps = 1; steps < max; steps++)
		{
			if (n & 1)
				n = (n << 1) + n + 1;
			else
				n >>= 1;

			if (n == 1)
			{
				stepsArray[i] = steps;
				return;
			}
		}
		stepsArray[i] = -1;
	}
}
